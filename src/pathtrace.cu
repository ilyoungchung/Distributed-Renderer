#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"

#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include <stream_compaction/efficient.h>

#define DI 0
#define DOF 0
#define SHOW_TIMING 0
#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

__host__ __device__ thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene *hst_scene = NULL;
static glm::vec3 *dev_image = NULL;
// TODO: static variables for device memory, scene/camera info, etc
// ...

static Camera *dev_camera = NULL;
static Geom *dev_geoms = NULL;
static int* dev_geoms_count = NULL;
static MeshGeom *dev_meshes = NULL;
static int *dev_meshes_count = NULL;
static Material *dev_materials = NULL;
static RenderState *dev_state = NULL;
static RayState *dev_rays_begin = NULL;
static RayState *dev_rays_end = NULL;
static int *dev_light_indices = NULL;
static int *dev_light_count = NULL;

//Initialise cuda memory
void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	//2D Pixel array to store image color
    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    //Copy Camera
    hipMalloc((void**)&dev_camera, sizeof(Camera));
    hipMemcpy(dev_camera, &hst_scene->state.camera, sizeof(Camera), hipMemcpyHostToDevice);

	//Copy geometry count
	int geom_count = hst_scene->geoms.size();
	hipMalloc((void**)&dev_geoms_count, sizeof(int));
	hipMemcpy(dev_geoms_count, &geom_count, sizeof(int), hipMemcpyHostToDevice);
	//Copy geometry
	hipMalloc((void**)&dev_geoms, geom_count * sizeof(Geom));
	hipMemcpy(dev_geoms, hst_scene->geoms.data(), geom_count * sizeof(Geom), hipMemcpyHostToDevice);

	copyMeshes();
	
	//Copy material
    hipMalloc((void**)&dev_materials, hst_scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, hst_scene->materials.data(), hst_scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    //Copy state
    hipMalloc((void**)&dev_state, sizeof(RenderState));
    hipMemcpy(dev_state, &hst_scene->state, sizeof(RenderState), hipMemcpyHostToDevice);

    //Allocate memory for rays
    hipMalloc((void**)&dev_rays_begin, pixelcount * sizeof(RayState));
//    hipMalloc((void**)&dev_rays_end, sizeof(RayState));

    //Copy Light Indices
    hipMalloc((void**)&dev_light_indices, hst_scene->state.lightIndices.size() * sizeof(int));
    hipMemcpy(dev_light_indices, hst_scene->state.lightIndices.data(), hst_scene->state.lightIndices.size() * sizeof(int), hipMemcpyHostToDevice);

    //Copy Light Count
    int lightCount = hst_scene->state.lightIndices.size();
    hipMalloc((void**)&dev_light_count, sizeof(int));
    hipMemcpy(dev_light_count, &lightCount, sizeof(int), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {

	hipFree(dev_image);
    // TODO: clean up the above static variables

    hipFree(dev_camera);
    hipFree(dev_geoms);
    hipFree(dev_geoms_count);
	hipFree(dev_meshes);
	hipFree(dev_meshes_count);
    hipFree(dev_materials);
    hipFree(dev_state);
    hipFree(dev_rays_begin);
//    hipFree(dev_rays_end);
    hipFree(dev_light_indices);
    hipFree(dev_light_count);

    checkCUDAError("pathtraceFree");
}

void copyMeshes()
{
	//Copy meshes count
	int mesh_count = hst_scene->meshGeoms.size();

	MeshGeom *allMeshes = new MeshGeom[mesh_count];

	hipMalloc((void**)&dev_meshes_count, sizeof(int));
	hipMemcpy(dev_meshes_count, &mesh_count, sizeof(int), hipMemcpyHostToDevice);

	for (int i = 0; i < mesh_count; ++i)
	{
		/*meshes[i] = hst_scene->meshGeoms[i];
		meshes[i].numVertices = hst_scene->meshGeoms[i].numVertices;*/
		MeshGeom meshes;

		meshes = hst_scene->meshGeoms[i];
		meshes.numVertices = hst_scene->meshGeoms[i].numVertices;

		glm::vec3 *triangles, *normals;

		hipMalloc(&triangles, meshes.numVertices * sizeof(glm::vec3));
		hipMalloc(&normals, meshes.numVertices * sizeof(glm::vec3));

		hipMemcpy(triangles, hst_scene->meshGeoms[i].triangles, meshes.numVertices * sizeof(glm::vec3), hipMemcpyHostToDevice);
		hipMemcpy(normals, hst_scene->meshGeoms[i].normals, meshes.numVertices * sizeof(glm::vec3), hipMemcpyHostToDevice);

		meshes.normals = normals;
		meshes.triangles = triangles;

		allMeshes[i] = meshes;
		/*meshes[i].normals = normals;
		meshes[i].triangles = triangles;*/
	}

	hipMalloc((void**)&dev_meshes, mesh_count * sizeof(MeshGeom));
	hipMemcpy(dev_meshes, allMeshes, mesh_count * sizeof(MeshGeom), hipMemcpyHostToDevice);
}

//Kernel function that gets all the ray directions
__global__ void kernGetRayDirections(Camera * camera, RayState* rays, int iter)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < camera->resolution.x && y < camera->resolution.y)
	{
		int index = x + (y * camera->resolution.x);

		//TODO : Tweak the random variable here if the image looks fuzzy
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(0, 0.005);

		//Find the ray direction
		float sx = float(x) / ((float) (camera->resolution.x) - 1.0f);
		float sy = float(y) / ((float) (camera->resolution.y) - 1.0f);

		glm::vec3 rayDir = (camera->M - (2.0f*sx - 1.0f + u01(rng)) * camera->H - (2.0f*sy - 1.0f + u01(rng)) * camera->V);
//		glm::vec3 rayDir = (camera->M - (2.0f*sx - 1.0f) * camera->H - (2.0f*sy - 1.0f) * camera->V);

		rayDir -= camera->position;
		rayDir = glm::normalize(rayDir);

		rays[index].ray.direction = rayDir;
		rays[index].ray.origin = camera->position;
		rays[index].isAlive = true;
		rays[index].rayColor = glm::vec3(0);
		rays[index].pixelIndex = index;
		rays[index].rayThroughPut = 1.0f;

//		printf("%d %d : %f %f %f\n", x, y, rayDir.x, rayDir.y, rayDir.z);
	}
}

//Kernel function that generates the Depth of field jitter
__global__ void kernJitterDOF(Camera * camera, RayState* rays, int iter)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < camera->resolution.x && y < camera->resolution.y)
	{
		int index = x + (y * camera->resolution.x);

		Ray &r = rays[index].ray;

		glm::vec3 intersectionPoint, normal;

		sphereIntersectionTest(camera->camSphere, r, intersectionPoint, normal);//, outside);

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(-0.5, 0.5);

		r.origin += glm::vec3(camera->aperture * u01(rng), camera->aperture * u01(rng), 0);
		r.direction = glm::normalize(intersectionPoint - r.origin);
	}
}


//Kernel function that performs one iteration of tracing the path.
__global__ void kernTracePath(Camera * camera, RayState *ray, Geom * geoms, int *geomCount, MeshGeom *meshGeoms, int *meshCount, int* lightIndices, int *lightCount, Material* materials, glm::vec3* image, int iter, int currDepth, int rayCount)
{
	 int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	 if (index < rayCount)
	 {
		 if(ray[index].isAlive)
		 {
			 glm::vec3 intersectionPoint = glm::vec3(0), normal = glm::vec3(0);
			 float min_t = FLT_MAX, t;
			 RayState &r = ray[index];
			 int nearestIndex = -1;
			 glm::vec3 nearestIntersectionPoint = glm::vec3(0), nearestNormal = glm::vec3(0);
//			 bool outside = false;

			 //Find geometry intersection
			 for(int i=0; i<(*geomCount); ++i)
			 {
				 if(geoms[i].type == CUBE)
				 {
					 t = boxIntersectionTest(geoms[i], r.ray, intersectionPoint, normal);//, outside);
				 }

				 else if(geoms[i].type == SPHERE)
				 {
					 t = sphereIntersectionTest(geoms[i], r.ray, intersectionPoint, normal);//, outside);
				 }

				 else if (geoms[i].type == MESH)
				 {
					 t = meshIntersectionTest(geoms[i], meshGeoms[geoms[i].meshid], r.ray, intersectionPoint, normal);//, outside);
				 }

				 if (t > 0 && t < min_t)//&& !outside)
				 {
					 min_t = t;
					 nearestIntersectionPoint = intersectionPoint;
					 nearestIndex = i;
					 nearestNormal = normal;
				 }
			 }

			 //If the nearest index remains unchanged, means no intersection and we can kill the ray.
			 if(nearestIndex == -1)
			 {
				 r.isAlive = false;

				 //Write the accumulated color for that pixel.
				 image[r.pixelIndex] += r.rayColor;
			 }

			 //else find the material color
			 else
			 {
				 //If light source
				 if(materials[geoms[nearestIndex].materialid].emittance >= 1)
				 {
					 //Light source, end ray here
					 r.isAlive = false;
					 
					 //If this is the primary ray, write the light color
					 if (currDepth == 0)
					 {
						 image[r.pixelIndex] += materials[geoms[nearestIndex].materialid].emittance
												* materials[geoms[nearestIndex].materialid].color;
					 }
					 //Else write the accumulated color
					 else
					 {
						 image[r.pixelIndex] += (r.rayColor);
					 }
				 }

				 else
				 {
					 thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, currDepth);

					 getRayColor(camera->position,
						 r,
						 nearestIntersectionPoint,
						 nearestNormal,
						 materials,
						 rng,
						 geoms,
						 nearestIndex,
						 geomCount,
						 meshGeoms,
						 meshCount,
						 lightIndices,
						 lightCount);
					 

					 /*scatterRay(camera->position,
								 r,
								 nearestIntersectionPoint,
								 nearestNormal,
								 materials[geoms[nearestIndex].materialid],
								 rng,
								 geoms,
								 nearestIndex,
								 lightIndices,
								 lightCount);*/

					 //TODO: Remove next line for path tracing
					 //image[r.pixelIndex] += r.rayColor;
				 }
			 }
		 }
	 }
}

__global__ void kernDirectLightPath(Camera * camera, RayState *ray, Geom * geoms, int * lightIndices, int* lightCount, glm::vec3* image, int iter, int currDepth, int rayCount)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < rayCount)
	{
		if(ray[index].isAlive)
		{
			glm::vec3 intersectionPoint, normal;
			float t;

			RayState &r = ray[index];
			int i;
			//bool outside;
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, currDepth);

			glm::vec3 pointOnLight = getRandomPointOnLight(geoms, lightIndices, lightCount, rng, i);

			r.ray.direction = glm::normalize(pointOnLight - r.ray.origin);
			t = sphereIntersectionTest(geoms[i], r.ray, intersectionPoint, normal);

			if(t > 0)
			{
				//Intersection with light, write the color
				image[r.pixelIndex] += r.rayColor;
				
				/*image[r.pixelIndex] += (r.rayColor
											 * materials[geoms[i].materialid].emittance
											 * materials[geoms[i].materialid].color);*/
			}
		}
	}
}


__global__ void kernWritePixels(Camera * camera, RayState *ray, glm::vec3* image, int rayCount)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < rayCount)
	{
		if (ray[index].isAlive)
		{
			image[ray[index].pixelIndex] += ray[index].rayColor;
		}
	}
}

__global__ void kernRussianRoullete(Camera * camera, RayState *ray, glm::vec3* image, int iter, int rayCount)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < rayCount)
	{
		if (ray[index].isAlive)
		{
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
			thrust::uniform_real_distribution<float> u01(0, 1.0f);
			
			if (ray[index].rayThroughPut < u01(rng))
			{
				ray[index].isAlive = false;
				image[ray[index].pixelIndex] += ray[index].rayColor;
			}
		}
	}
}

struct isDead
{
	__host__ __device__ bool  operator()(const RayState r)
	{
		return (!r.isAlive);
	}
};


/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */

void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    const int blockSideLength = 8;
    dim3 blockSize(blockSideLength, blockSideLength);
    dim3 blocksPerGrid(
            (cam.resolution.x + blockSize.x - 1) / blockSize.x,
            (cam.resolution.y + blockSize.y - 1) / blockSize.y);

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    // * For each depth:
    //   * Compute one new (ray, color) pair along each path (using scatterRay).
    //     Note that many rays will terminate by hitting a light or hitting
    //     nothing at all. You'll have to decide how to represent your path rays
    //     and how you'll mark terminated rays.
    //   * Add all of the terminated rays' results into the appropriate pixels.
    //   * Stream compact away all of the terminated paths.
    //     You may use your implementation or `thrust::remove_if` or its
    //     cousins.
    // * Finally, handle all of the paths that still haven't terminated.
    //   (Easy way is to make them black or background-colored.)

    // TODO: perform one iteration of path tracing

    //Setup initial rays
    kernGetRayDirections<<< blocksPerGrid, blockSize>>>(dev_camera, dev_rays_begin, iter);

    //Jitter rays as per Depth of field
    if(DOF)
    {
    	kernJitterDOF<<<blocksPerGrid, blockSize>>>(dev_camera, dev_rays_begin, iter);
    }

    dev_rays_end = dev_rays_begin + pixelcount;
    int rayCount = pixelcount;
    int numBlocks, numThreads = 128;

    numBlocks = (rayCount + numThreads - 1) / numThreads;
	int i;

    for(int i=0; (i<traceDepth && rayCount > 0); ++i)	//For Path Tracing
	//for (int i = 0; i<1; ++i)							//For DI
    {
//    	hipEvent_t start, stop;
//    	hipEventCreate(&start);
//    	hipEventCreate(&stop);
//    	hipEventRecord(start);

    	//Take one step, should make dead rays as false
    	kernTracePath<<<numBlocks, numThreads>>>(dev_camera, dev_rays_begin, dev_geoms, dev_geoms_count, dev_meshes, dev_meshes_count, dev_light_indices, dev_light_count, dev_materials, dev_image, iter, i, rayCount);
		checkCUDAError("pathtrace step");

		//If currDepth is > 2, play russian roullete
		if (i > 2)
		{
			kernRussianRoullete << <numBlocks, numThreads >> >(dev_camera, dev_rays_begin, dev_image, iter, rayCount);
		}

		// Compact rays, dev_rays_end points to the new end
		dev_rays_end = thrust::remove_if(thrust::device, dev_rays_begin, dev_rays_end, isDead());
    	rayCount = dev_rays_end - dev_rays_begin;

		//Calculate new number of blocks
    	numBlocks = (rayCount + numThreads - 1) / numThreads;

//    	hipEventRecord(stop);
//    	hipEventSynchronize(stop);
//    	float milliseconds = 0;
//    	hipEventElapsedTime(&milliseconds, start, stop);
//    	if(SHOW_TIMING)
//    		std::cout<</*"Iter : "<<iter<<" Depth : "<<i<<" Total time in milliseconds : "<<*/milliseconds<<std::endl;
    }

	//std::cout << i << std::endl;
	if (rayCount > 0)
	{
		kernWritePixels << <numBlocks, numThreads >> >(dev_camera, dev_rays_begin, dev_image, rayCount);
	}

    //Direct Illumination
    if(DI && rayCount > 0)
    {
    	kernDirectLightPath<<<numBlocks, numThreads>>>(dev_camera, dev_rays_begin, dev_geoms, dev_light_indices, dev_light_count, dev_image, iter, traceDepth, rayCount);
    }

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid, blockSize>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
