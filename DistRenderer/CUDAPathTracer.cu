#include "hip/hip_runtime.h"
#include "CUDAPathTracer.h"

CUDAPathTracer::CUDAPathTracer(){
	active = false;
}

CUDAPathTracer::~CUDAPathTracer(){
	pathtraceFree();
};

int CUDAPathTracer::getPixelCount(){
	return pixelcount;
}

void CUDAPathTracer::pathtraceInit(Scene* scene, int rendererNo_, int totalRenderer_)
{
	active = true;

	hst_scene = scene;
	const Camera &cam = hst_scene->state.camera;

	rendererNo = rendererNo_;
	totalRenderer = totalRenderer_;

	width = cam.resolution.x;
	height = cam.resolution.y;

	pixelcount = (width * height) / totalRenderer;
	if (width * height % totalRenderer > rendererNo)
		pixelcount++;
		
	std::cout << "Pixel to be rendered: " << pixelcount << std::endl;

	//2D Pixel array to store image color
	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	checkCUDAError("pathtraceInit");
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));
	checkCUDAError("pathtraceInit");

	//Copy Camera
	hipMalloc((void**)&dev_camera, sizeof(Camera));
	hipMemcpy(dev_camera, &hst_scene->state.camera, sizeof(Camera), hipMemcpyHostToDevice);

	//Copy geometry count
	int geom_count = hst_scene->geoms.size();
	hipMalloc((void**)&dev_geoms_count, sizeof(int));
	hipMemcpy(dev_geoms_count, &geom_count, sizeof(int), hipMemcpyHostToDevice);

	//Copy geometry
	hipMalloc((void**)&dev_geoms, geom_count * sizeof(Geom));
	hipMemcpy(dev_geoms, hst_scene->geoms.data(), geom_count * sizeof(Geom), hipMemcpyHostToDevice);

	copyMeshes();

	//Copy material
	hipMalloc((void**)&dev_materials, hst_scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, hst_scene->materials.data(), hst_scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	//Copy state
	hipMalloc((void**)&dev_state, sizeof(RenderState));
	hipMemcpy(dev_state, &hst_scene->state, sizeof(RenderState), hipMemcpyHostToDevice);

	//Allocate memory for rays
	hipMalloc((void**)&dev_rays_begin, pixelcount * sizeof(RayState));
	//    hipMalloc((void**)&dev_rays_end, sizeof(RayState));

	//Copy Light Indices
	hipMalloc((void**)&dev_light_indices, hst_scene->state.lightIndices.size() * sizeof(int));
	hipMemcpy(dev_light_indices, hst_scene->state.lightIndices.data(), hst_scene->state.lightIndices.size() * sizeof(int), hipMemcpyHostToDevice);

	//Copy Light Count
	int lightCount = hst_scene->state.lightIndices.size();
	hipMalloc((void**)&dev_light_count, sizeof(int));
	hipMemcpy(dev_light_count, &lightCount, sizeof(int), hipMemcpyHostToDevice);

	checkCUDAError("pathtraceInit");
}

void CUDAPathTracer::pathtraceFree(){
	hipFree(dev_image);
	// TODO: clean up the above static variables

	hipFree(dev_camera);
	hipFree(dev_geoms);
	hipFree(dev_geoms_count);
	hipFree(dev_meshes);
	hipFree(dev_meshes_count);
	hipFree(dev_materials);
	hipFree(dev_state);
	hipFree(dev_rays_begin);
	//    hipFree(dev_rays_end);
	hipFree(dev_light_indices);
	hipFree(dev_light_count);

	active = false;
}

const std::vector<glm::vec3>& CUDAPathTracer::getPixels(){
	return hst_scene->state.image;
}

bool CUDAPathTracer::isActive(){
	return active;
}

void CUDAPathTracer::setCamera(float theta, float phi, glm::vec3 cammove){
	Camera &cam = hst_scene->state.camera; 
	glm::vec3 v = cam.view;
	glm::vec3 u = cam.up;
	glm::vec3 r = glm::cross(v, u);
	glm::mat4 rotmat = glm::rotate(theta, r) * glm::rotate(phi, u);
	cam.view = glm::vec3(rotmat * glm::vec4(v, 0.f));
	cam.up = glm::vec3(rotmat * glm::vec4(u, 0.f));
	cam.position += cammove.x * r + cammove.y * u + cammove.z * v;

	hst_scene->configureCamera();
}

void CUDAPathTracer::pathtrace(uchar4 *pbo, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera &cam = hst_scene->state.camera;

	int blockSize = 64;
	int blocksPerGrid = glm::ceil(pixelcount / (float)blockSize);

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	// * For each depth:
	//   * Compute one new (ray, color) pair along each path (using scatterRay).
	//     Note that many rays will terminate by hitting a light or hitting
	//     nothing at all. You'll have to decide how to represent your path rays
	//     and how you'll mark terminated rays.
	//   * Add all of the terminated rays' results into the appropriate pixels.
	//   * Stream compact away all of the terminated paths.
	//     You may use your implementation or `thrust::remove_if` or its
	//     cousins.
	// * Finally, handle all of the paths that still haven't terminated.
	//   (Easy way is to make them black or background-colored.)

	// TODO: perform one iteration of path tracing

	//Setup initial rays
	kernGetRayDirections << < blocksPerGrid, blockSize >> >(dev_camera, dev_rays_begin, iter,
		rendererNo, totalRenderer);
	checkCUDAError("kernGetRayDirections");

	//Jitter rays as per Depth of field
	if (DOF)
	{
		kernJitterDOF << <blocksPerGrid, blockSize >> >(dev_camera, dev_rays_begin, iter,
			rendererNo, totalRenderer);
		checkCUDAError("kernJitterDOF");
	}

	dev_rays_end = dev_rays_begin + pixelcount;
	int rayCount = pixelcount;
	int numBlocks, numThreads = 128;

	for (int i = 0; (i<traceDepth && rayCount > 0); ++i)	//For Path Tracing
		//for (int i = 0; i<1; ++i)							//For DI
	{
		//Calculate new number of blocks
		numBlocks = (rayCount + numThreads - 1) / numThreads;

		//    	hipEvent_t start, stop;
		//    	hipEventCreate(&start);
		//    	hipEventCreate(&stop);
		//    	hipEventRecord(start);

		//Take one step, should make dead rays as false
		//std::cout << i << " tracedepth: " << traceDepth << " / raycount: " << rayCount << std::endl;
		kernTracePath << <numBlocks, numThreads >> >(dev_camera, dev_rays_begin, dev_geoms, dev_geoms_count, dev_meshes, dev_meshes_count, dev_light_indices, dev_light_count, dev_materials, dev_image, iter, i, rayCount);
		checkCUDAError("kernTracePath");

		//If currDepth is > 2, play russian roullete
		if (i > 2)
		{
			kernRussianRoullete << <numBlocks, numThreads >> >(dev_camera, dev_rays_begin, dev_image, iter, rayCount);
			checkCUDAError("kernRussianRoullete");
		}

		// Compact rays, dev_rays_end points to the new end
		dev_rays_end = thrust::remove_if(thrust::device, dev_rays_begin, dev_rays_end, isDead());
		rayCount = dev_rays_end - dev_rays_begin;

		//    	hipEventRecord(stop);
		//    	hipEventSynchronize(stop);
		//    	float milliseconds = 0;
		//    	hipEventElapsedTime(&milliseconds, start, stop);
		//    	if(SHOW_TIMING)
		//    		std::cout<</*"Iter : "<<iter<<" Depth : "<<i<<" Total time in milliseconds : "<<*/milliseconds<<std::endl;
	}

	//std::cout << i << std::endl;
	if (rayCount > 0)
	{
		kernWritePixels << <numBlocks, numThreads >> >(dev_camera, dev_rays_begin, dev_image, rayCount);
		checkCUDAError("kernWritePixels");
	}

	//Direct Illumination
	if (DI && rayCount > 0)
	{
		kernDirectLightPath << <numBlocks, numThreads >> >(dev_camera, dev_rays_begin, dev_geoms, dev_light_indices, dev_light_count, dev_image, iter, traceDepth, rayCount);
		checkCUDAError("kernDirectLightPath");
	}

	// Send results to OpenGL buffer for rendering
	if (pbo != nullptr){
		sendImageToPBO << <blocksPerGrid, blockSize >> >(pbo, cam.resolution, iter, dev_image, rendererNo, totalRenderer);
		checkCUDAError("sendImageToPBO");
	}

	// TODO: Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}

void CUDAPathTracer::saveImage(std::string title, int iteration) {
	float samples = iteration;
	// output image file
	image img(width, height);

	for (int ptr = 0; ptr < pixelcount; ptr++) {
		int index = rendererNo + ptr * totalRenderer;
		int y = index / width;
		int x = index - (y * width);

		glm::vec3 pix = hst_scene->state.image[ptr];
		img.setPixel(width - 1 - x, y, glm::vec3(pix) / samples);
	}

	std::string filename = hst_scene->state.imageName;
	filename = filename + "." + title + "." + std::to_string(iteration) + "samp";

	// CHECKITOUT
	img.savePNG(filename);
	//img.saveHDR(filename);  // Save a Radiance HDR file
}

void CUDAPathTracer::copyMeshes()
{
	//Copy meshes count
	int mesh_count = hst_scene->meshGeoms.size();

	MeshGeom *allMeshes = new MeshGeom[mesh_count];

	hipMalloc((void**)&dev_meshes_count, sizeof(int));
	hipMemcpy(dev_meshes_count, &mesh_count, sizeof(int), hipMemcpyHostToDevice);

	for (int i = 0; i < mesh_count; ++i)
	{
		/*meshes[i] = hst_scene->meshGeoms[i];
		meshes[i].numVertices = hst_scene->meshGeoms[i].numVertices;*/
		MeshGeom meshes;

		meshes = hst_scene->meshGeoms[i];
		meshes.numVertices = hst_scene->meshGeoms[i].numVertices;

		glm::vec3 *triangles, *normals;

		hipMalloc(&triangles, meshes.numVertices * sizeof(glm::vec3));
		hipMalloc(&normals, meshes.numVertices * sizeof(glm::vec3));

		hipMemcpy(triangles, hst_scene->meshGeoms[i].triangles, meshes.numVertices * sizeof(glm::vec3), hipMemcpyHostToDevice);
		hipMemcpy(normals, hst_scene->meshGeoms[i].normals, meshes.numVertices * sizeof(glm::vec3), hipMemcpyHostToDevice);

		meshes.normals = normals;
		meshes.triangles = triangles;

		allMeshes[i] = meshes;
		/*meshes[i].normals = normals;
		meshes[i].triangles = triangles;*/
	}

	hipMalloc((void**)&dev_meshes, mesh_count * sizeof(MeshGeom));
	hipMemcpy(dev_meshes, allMeshes, mesh_count * sizeof(MeshGeom), hipMemcpyHostToDevice);
}

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
	exit(EXIT_FAILURE);
}

__host__ __device__ thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image,
	int rendererNo, int totalRenderer)
{
	int ptr = (blockIdx.x * blockDim.x) + threadIdx.x;
	int index = rendererNo + ptr * totalRenderer;

	if (index < resolution.x * resolution.y) {
		glm::vec3 pix = image[ptr];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

//Kernel function that gets all the ray directions
__global__ void kernGetRayDirections(Camera * camera, RayState* rays, int iter,
	int rendererNo, int totalRenderer)
{
	int ptr = (blockIdx.x * blockDim.x) + threadIdx.x;
	int index = rendererNo + ptr * totalRenderer;

	if (index < camera->resolution.x * camera->resolution.y)
	{
		//TODO : Tweak the random variable here if the image looks fuzzy
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(0, 0.005);

		//Find the ray direction
		int p = index / camera->resolution.x;
		float sy = float(p) / ((float)(camera->resolution.y) - 1.0f);

		p = index - (p * camera->resolution.x);
		float sx = float(p) / ((float)(camera->resolution.x) - 1.0f);

		glm::vec3 rayDir = (camera->M - (2.0f*sx - 1.0f + u01(rng)) * camera->H - (2.0f*sy - 1.0f + u01(rng)) * camera->V);
		//		glm::vec3 rayDir = (camera->M - (2.0f*sx - 1.0f) * camera->H - (2.0f*sy - 1.0f) * camera->V);

		rayDir -= camera->position;
		rayDir = glm::normalize(rayDir);

		rays[ptr].ray.direction = rayDir;
		rays[ptr].ray.origin = camera->position;
		rays[ptr].isAlive = true;
		rays[ptr].rayColor = glm::vec3(0);
		rays[ptr].pixelIndex = ptr;
		rays[ptr].rayThroughPut = 1.0f;

		//		printf("%d %d : %f %f %f\n", x, y, rayDir.x, rayDir.y, rayDir.z);
	}
}

//Kernel function that generates the Depth of field jitter
__global__ void kernJitterDOF(Camera * camera, RayState* rays, int iter,
	int rendererNo, int totalRenderer)
{
	int ptr = (blockIdx.x * blockDim.x) + threadIdx.x;
	int index = rendererNo + ptr * totalRenderer;

	if (index < camera->resolution.x * camera->resolution.y)
	{
		Ray &r = rays[ptr].ray;

		glm::vec3 intersectionPoint, normal;

		sphereIntersectionTest(camera->camSphere, r, intersectionPoint, normal);//, outside);

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(-0.5, 0.5);

		r.origin += glm::vec3(camera->aperture * u01(rng), camera->aperture * u01(rng), 0);
		r.direction = glm::normalize(intersectionPoint - r.origin);
	}
}


//Kernel function that performs one iteration of tracing the path.
__global__ void kernTracePath(Camera * camera, RayState *ray, Geom * geoms, int *geomCount, MeshGeom *meshGeoms, int *meshCount, int* lightIndices, int *lightCount, Material* materials, glm::vec3* image, int iter, int currDepth, int rayCount)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < rayCount)
	{
		if (ray[index].isAlive)
		{
			glm::vec3 intersectionPoint = glm::vec3(0), normal = glm::vec3(0);
			float min_t = FLT_MAX, t;
			RayState &r = ray[index];
			int nearestIndex = -1;
			glm::vec3 nearestIntersectionPoint = glm::vec3(0), nearestNormal = glm::vec3(0);
			bool outside = false;

			//Find geometry intersection
			for (int i = 0; i<(*geomCount); ++i)
			{
				if (geoms[i].type == CUBE)
				{
					t = boxIntersectionTest(geoms[i], r.ray, intersectionPoint, normal, outside);
				}

				else if (geoms[i].type == SPHERE)
				{
					t = sphereIntersectionTest(geoms[i], r.ray, intersectionPoint, normal, outside);
				}

				else if (geoms[i].type == MESH)
				{
					t = meshIntersectionTest(geoms[i], meshGeoms[geoms[i].meshid], r.ray, intersectionPoint, normal, outside);
				}

				if (t > 0 && t < min_t)
				{
					min_t = t;
					nearestIntersectionPoint = intersectionPoint;
					nearestIndex = i;
					nearestNormal = normal;
				}
			}

			//If the nearest index remains unchanged, means no intersection and we can kill the ray.
			if (nearestIndex == -1)
			{
				r.isAlive = false;

				//Write the accumulated color for that pixel.
				image[r.pixelIndex] += r.rayColor;
			}

			//else find the material color
			else
			{
				//If light source
				if (materials[geoms[nearestIndex].materialid].emittance >= 1)
				{
					//Light source, end ray here
					r.isAlive = false;

					//If this is the primary ray, write the light color
					if (currDepth == 0)
					{
						image[r.pixelIndex] += materials[geoms[nearestIndex].materialid].emittance
							* materials[geoms[nearestIndex].materialid].color;
					}
					//Else write the accumulated color
					else
					{
						image[r.pixelIndex] += (r.rayColor);
					}
				}

				else
				{
					thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, currDepth);

					getRayColor(camera->position,
						r,
						nearestIntersectionPoint,
						nearestNormal,
						outside,
						materials,
						rng,
						geoms,
						nearestIndex,
						geomCount,
						meshGeoms,
						meshCount,
						lightIndices,
						lightCount);


					/*scatterRay(camera->position,
					r,
					nearestIntersectionPoint,
					nearestNormal,
					materials[geoms[nearestIndex].materialid],
					rng,
					geoms,
					nearestIndex,
					lightIndices,
					lightCount);*/

					//TODO: Remove next line for path tracing
					//image[r.pixelIndex] += r.rayColor;
				}
			}
		}
	}
}

__global__ void kernDirectLightPath(Camera * camera, RayState *ray, Geom * geoms, int * lightIndices, int* lightCount, glm::vec3* image, int iter, int currDepth, int rayCount)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < rayCount)
	{
		if (ray[index].isAlive)
		{
			glm::vec3 intersectionPoint, normal;
			float t;

			RayState &r = ray[index];
			int i;
			//bool outside;
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, currDepth);

			glm::vec3 pointOnLight = getRandomPointOnLight(geoms, lightIndices, lightCount, rng, i);

			r.ray.direction = glm::normalize(pointOnLight - r.ray.origin);
			t = sphereIntersectionTest(geoms[i], r.ray, intersectionPoint, normal);

			if (t > 0)
			{
				//Intersection with light, write the color
				image[r.pixelIndex] += r.rayColor;

				/*image[r.pixelIndex] += (r.rayColor
				* materials[geoms[i].materialid].emittance
				* materials[geoms[i].materialid].color);*/
			}
		}
	}
}

// Write the color of the rays that are still alive
__global__ void kernWritePixels(Camera * camera, RayState *ray, glm::vec3* image, int rayCount)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < rayCount)
	{
		if (ray[index].isAlive)
		{
			image[ray[index].pixelIndex] += ray[index].rayColor;
		}
	}
}

//Kill rays based on the throughput and a random number generated between 0 and 1
__global__ void kernRussianRoullete(Camera * camera, RayState *ray, glm::vec3* image, int iter, int rayCount)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < rayCount)
	{
		if (ray[index].isAlive)
		{
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
			thrust::uniform_real_distribution<float> u01(0, 1.0f);

			if (ray[index].rayThroughPut < u01(rng))
			{
				ray[index].isAlive = false;
				image[ray[index].pixelIndex] += ray[index].rayColor;
			}
		}
	}
}